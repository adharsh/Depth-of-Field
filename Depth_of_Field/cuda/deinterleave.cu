#include "hip/hip_runtime.h"
#include "deinterleave.cuh"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include "summed_area_table.cuh"

__global__ void deinterleave_kernel(unsigned char* image, unsigned int NxN, unsigned int* r, unsigned int* g, unsigned int* b) 
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < NxN) {
		
		unsigned char* pixelOffset = image + tid * 3;
		r[tid] = pixelOffset[0];
		g[tid] = pixelOffset[1];
		b[tid] = pixelOffset[2];

		tid += blockDim.x * gridDim.x;
	}
}

void deinterleave(unsigned char* image, unsigned int NxN, unsigned int** r, unsigned int** g, unsigned int** b) 
{
	
	//input: img
	//output: r, g, b

	unsigned char* dev_img;
	hipMalloc((void**)&dev_img, 3 * NxN * sizeof(char));

	unsigned int* dev_r, *dev_g, *dev_b;
	hipMalloc((void**)&dev_r, NxN * sizeof(int));
	hipMalloc((void**)&dev_g, NxN * sizeof(int));
	hipMalloc((void**)&dev_b, NxN * sizeof(int));
	
	hipMemcpy(dev_img, image, 3 * NxN * sizeof(char), hipMemcpyHostToDevice);

	deinterleave_kernel << <512, 512>> > (dev_img, NxN, dev_r, dev_g, dev_b);
	
	hipMemcpy(*r, dev_r, NxN * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(*g, dev_g, NxN * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(*b, dev_b, NxN * sizeof(int), hipMemcpyDeviceToHost);
	
	/*for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			std::cout << *r[i * 512 + j] << " ";
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;*/

	hipFree(dev_r);
	hipFree(dev_g);
	hipFree(dev_b);
}


//int main()
//{
//	//input: img
//	//output: r, g, b
//
//	unsigned int N = 4; //N x N image
//
//	unsigned char* img = (unsigned char*)malloc(3 * N * N * sizeof(char));
//	for (int i = 0; i < 3 * N * N; i++)
//		img[i] = 1;
//	unsigned int* r = (unsigned int*)malloc(N * N * sizeof(int));
//	unsigned int* g = (unsigned int*)malloc(N * N * sizeof(int));
//	unsigned int* b = (unsigned int*)malloc(N * N *sizeof(int));
//
//	deinterleave(img, N * N, &r, &g, &b);
//
//	/*for (int i = 0; i < N; i++)
//	{
//		std::cout << r[i] << " " << g[i] << " " << b[i] << " ";
//	}*/
//
//	summed_area_table(g, N);
//
//	for (int i = 0; i < N; i++)
//	{
//		for (int j = 0; j < N; j++)
//		{
//			std::cout << g[i * N + j] << " ";
//		}
//		std::cout << std::endl;
//	}
//
//
//	return 0;
//}
